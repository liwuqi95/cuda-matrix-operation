#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>

// time stamp function in seconds
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec / 1000000 + tv.tv_sec;
}

// host side matrix addition
void h_inverse(float *A, float *B, int nx, int ny) {
    for (int i = 0; i < ny; i++)
        for (int j = 0; j < nx; j++)
            B[j * ny + i] = A[i * nx + j];
}

// device-side matrix addition
__global__ void f_inverse(float *A, float *B, int nx, int ny, bool reverse) {
    __shared__ float sdata[32][33];

    int xBlock, yBlock, threadIndex;


    if (reverse) {
        int xBlock = blockIdx.y * blockDim.y;
        int yBlock = blockIdx.x * blockDim.x * 32;
        threadIndex = threadIdx.y;
    } else {
        int xBlock = blockIdx.x * blockDim.x;
        int yBlock = blockIdx.y * blockDim.y * 32;
        threadIndex = threadIdx.x;
    }


    int ix = xBlock + threadIndex;
    int iy = yBlock;


    int x, y;

    for (int i = 0; i < 32; i++) {
        x = ix;
        y = iy + i;
        if (x < nx && y < ny)
            sdata[i][threadIndex] = A[y * nx + x];
    }

    __syncthreads();

    ix = yBlock + threadIndex;
    iy = xBlock;


    for (int i = 0; i < 32; i++) {
        x = ix;
        y = iy + i;
        if (x < ny && y < nx)
            B[x + y * ny] = sdata[threadIndex][i];
    }
}

int main(int argc, char *argv[]) {
    // get program arguments
    if (argc != 3) {
        printf("Error: wrong number of args\n");
        exit(1);
    }


    int nx = atoi(argv[1]); // should check validity
    int ny = atoi(argv[2]); // should check validity

    int noElems = nx * ny;
    int bytes = noElems * sizeof(float);
    // but you may want to pad the matrices…

    // alloc memory host-side
    float *h_A = (float *) malloc(bytes);

    float *h_hR = (float *) malloc(bytes); // host result
    float *h_dR = (float *) malloc(bytes); // gpu result

    hipHostRegister(h_A, bytes, 0);
    hipHostRegister(h_dR, bytes, 0);

    // init matrices with random data

    int i, j;
    for (i = 0; i < ny; i++)
        for (j = 0; j < nx; j++)
            h_A[i * nx + j] = rand();



    // alloc memory dev-side
    float *d_A, *d_R;
    hipMalloc((void **) &d_A, bytes);
    hipMalloc((void **) &d_R, bytes);

    double timeStampA = getTimeStamp();
    //transfer data to dev
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);


    // invoke Kernel
    dim3 block(32, 1);

    bool reverse = ny > nx;

    if (reverse)
        dim3 grid((ny + block.y * 32 - 1) / (block.y * 32), (nx + block.x - 1) / block.x);
    else
        dim3 grid((nx + block.x - 1) / block.x, (ny + block.y * 32 - 1) / (block.y * 32));

    f_inverse << < grid, block >> > (d_A, d_R, nx, ny, reverse);
    hipDeviceSynchronize();

    //copy data back
    hipMemcpy(h_dR, d_R, bytes, hipMemcpyDeviceToHost);

    double timeStampD = getTimeStamp();
    // free GPU resources
    hipFree(d_A);
    hipFree(d_R);
    hipDeviceReset();

    // check result
    h_inverse(h_A, h_hR, nx, ny);

    bool correct = true;

    for (i = 0; i < nx * ny; i++)
        if (h_hR[i] != h_dR[i]) {
            correct = false;
            break;
        }

    if (!correct)
        printf("Error: Result Incorrect!\n");

    hipHostUnregister(h_A);
    hipHostUnregister(h_dR);


    // print out results
    printf("%.6f \n", timeStampD - timeStampA);
}
