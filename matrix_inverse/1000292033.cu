#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>

// time stamp function in seconds
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec / 1000000 + tv.tv_sec;
}

// host side matrix addition
void h_inverse(float *A, float *B, int nx, int ny) {
    for (int i = 0; i < ny; i++)
        for (int j = 0; j < nx; j++)
            B[j * ny + i] = A[i * nx + j];
}

// device-side matrix addition
__global__ f_inverse(float *A, float *B, int nx, int ny) {
    int ix = threadId.x + bloackId.x * blockDim.x;
    int iy = threadId.y + bloackId.y * blockDim.y;
    int idx = iy * nx + ix;
    int n_idx = ix * ny + iy;
    if ((ix < nx) && (iy < ny))
        B[n_idx] = A[idx];
}

int main(int argc, char *argv[]) {
    // get program arguments
    if (argc != 3) {
        printf("Error: wrong number of args\n");
        exit(1);
    }


    int nx = atoi(argv[1]); // should check validity
    int ny = atoi(argv[2]); // should check validity

    int noElems = nx * ny;
    int bytes = noElems * sizeof(float);
    // but you may want to pad the matrices…

    // alloc memory host-side
    float *h_A = (float *) malloc(bytes);

    float *h_hR = (float *) malloc(bytes); // host result
    float *h_dR = (float *) malloc(bytes); // gpu result

    hipHostRegister(h_A, bytes, 0);
    hipHostRegister(h_dR, bytes, 0);

    // init matrices with random data

    int i, j, index;
    for (i = 0; i < ny; i++)
        for (j = 0; j < nx; j++)
            h_A[i * nx + j] = rand();



    // alloc memory dev-side
    float *d_A, *d_R;
    hipMalloc((void **) &d_A, bytes);
    hipMalloc((void **) &d_R, bytes);

    double timeStampA = getTimeStamp();
    //transfer data to dev
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);


    // invoke Kernel
    dim3 block(1024, 1);
    dim3 grid((noElems + 1023) / 1024);

    f_addmat << < grid, block >> > (d_A, d_R, nx, ny);
    hipDeviceSynchronize();

    //copy data back
    hipMemcpy(h_dR, d_R, bytes, hipMemcpyDeviceToHost);
    double timeStampD = getTimeStamp();

    // free GPU resources
    hipFree(d_A);
    hipFree(d_R);
    hipDeviceReset();

    // check result
    h_inverse(h_A, h_hR, nx, ny);

    bool correct = true;

    for (i = 0; i < nx * ny; i++)
        if (h_hC[i] != h_dC[i]) {
            correct = false;
            break;
        }

    if (!correct)
        printf("Error: Result Incorrect!\n");

    hipHostUnregister(h_A);
    hipHostUnregister(h_R);


    // print out results
    printf("%.6f \n", timeStampD - timeStampA);
}
