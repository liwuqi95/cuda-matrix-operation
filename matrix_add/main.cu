#include "hip/hip_runtime.h"
#include <sys/time.h>

// time stamp function in seconds
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec / 1000000 + tv.tv_sec;
}

// host side matrix addition
void h_addmat(float *A, float *B, float *C, int nx, int ny) {}

void initDataA(float *M, int num) {

    int i = 0;
    for (i = 0; i < num; i++) {
        M[i] = ((float) rand() / (float) (RAND_MAX)))
    }

}

// device-side matrix addition
__global__ void f_addmat(float *A, float *B, float *C, int nx, int ny) {
    // kernel code might look something like this
    // but you may want to pad the matrices and index into them accordingly
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = iy * ny + ix;
    if ((ix < nx) && (iy < ny))
        C[idx] = A[idx] + B[idx];
}

int main(int argc, char *argv[]) {
    // get program arguments
    if (argc != 3) {
        printf("Error: wrong number of args\n");
        exit(1);
    }
    int nx = atoi(argv[2]); // should check validity
    int ny = atoi(argv[3]); // should check validity
    int noElems = nx * ny;
    int bytes = noElems * sizeof(float);
    // but you may want to pad the matrices…

    // alloc memory host-side
    float *h_A = (float *) malloc(bytes);
    float *h_B = (float *) malloc(bytes);
    float *h_hC = (float *) malloc(bytes); // host result
    float *h_dC = (float *) malloc(bytes); // gpu result

    // init matrices with random data
    initDataA(h_A, noElems);

    //init A
    int i, j;
    for (i = 0; i < nx; i++)
        for (j = 0; j < ny; j++) {
            h_A[i][j] = (float) (i + j) / 3.0;
            h_B[i][j] = (float) 3.14 * (i + j);
        }


    // alloc memory dev-side
    float *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, bytes);
    hipMalloc((void **) &d_B, bytes);
    hipMalloc((void **) &d_C, bytes);

    double timeStampA = getTimeStamp();
    //transfer data to dev
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    // note that the transfers would be twice as fast if h_A and h_B
    // matrices are pinned
    double timeStampB = getTimeStamp();

    // invoke Kernel
    dim3 block(32, 32); // you will want to configure this
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    f_addmat << < grid, block >> > (d_A, d_B, d_C, nx, ny);
    hipDeviceSynchronize();

    double timeStampC = getTimeStamp();
    //copy data back
    hipMemcpy(h_dC, d_C, bytes, hipMemcpyDeviceToHost);
    double timeStampD = getTimeStamp();

    // free GPU resources
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceReset();

    // check result
    h_addmat(h_A, h_B, h_hC, nx, ny);
    // h_dC == h+hC???
    // print out results
}
